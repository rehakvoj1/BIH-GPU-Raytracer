﻿#pragma once
#include <hip/hip_runtime.h>

class Managed {
public:
    void* operator new( size_t len ) {
        void* ptr;
        hipMallocManaged( &ptr, len );
        hipDeviceSynchronize();
        return ptr;
    }

    void operator delete( void* ptr ) {
        hipDeviceSynchronize();
        hipFree( ptr );
    }
};