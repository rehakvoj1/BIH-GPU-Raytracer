﻿#include "Triangle.h"

__device__ bool Triangle::Hit( const Ray& r, float tmin, float tmax, HitRecord& rec ) const {
    glm::vec3 v1v2 = v2.Position - v1.Position;
    glm::vec3 v1v3 = v3.Position - v1.Position;

    glm::vec3 pvec = glm::cross( r.Direction(), v1v3 );

    float det = glm::dot( v1v2, pvec );

    if ( det < 0.000001 )
        return false;

    float invDet = 1.0 / det;

    glm::vec3 tvec = r.Origin() - v1.Position;

    float u = dot( tvec, pvec ) * invDet;

    if ( u < 0 || u > 1 )
        return false;

    glm::vec3 qvec = glm::cross( tvec, v1v2 );

    float v = glm::dot( r.Direction(), qvec ) * invDet;

    if ( v < 0 || u + v > 1 )
        return false;

    return true;
}
